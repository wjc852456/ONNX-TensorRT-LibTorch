#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaWarp.h"


// gpuIntrinsicWarp
template<typename T>
__global__ void gpuIntrinsicWarp( T* input, T* output, int width, int height,
						    float2 focalLength, float2 principalPoint, float k1, float k2, float p1, float p2)
{
	const int2 uv_out = make_int2(blockDim.x * blockIdx.x + threadIdx.x,
				               blockDim.y * blockIdx.y + threadIdx.y);
						   
	if( uv_out.x >= width || uv_out.y >= height )
		return;
	
	const float u = uv_out.x;
	const float v = uv_out.y;

	const float _fx = 1.0f / focalLength.x;
	const float _fy = 1.0f / focalLength.y;
	
	const float y      = (v - principalPoint.y)*_fy;
	const float y2     = y*y;
	const float _2p1y  = 2.0*p1*y;
	const float _3p1y2 = 3.0*p1*y2;
	const float p2y2   = p2*y2;

	const float x  = (u - principalPoint.x)*_fx;
	const float x2 = x*x;
	const float r2 = x2 + y2;
	const float d  = 1.0 + (k1 + k2*r2)*r2;
	const float _u = focalLength.x*(x*(d + _2p1y) + p2y2 + (3.0*p2)*x2) + principalPoint.x;
	const float _v = focalLength.y*(y*(d + (2.0*p2)*x) + _3p1y2 + p1*x2) + principalPoint.y;

	const int2 uv_in = make_int2( _u, _v );
	
	if( uv_in.x >= width || uv_in.y >= height || uv_in.x < 0 || uv_in.y < 0 )
		return;

	output[uv_out.y * width + uv_out.x] = input[uv_in.y * width + uv_in.x];
} 


// cudaWarpIntrinsic
hipError_t cudaWarpIntrinsic( uchar4* input, uchar4* output, uint32_t width, uint32_t height,
						 const float2& focalLength, const float2& principalPoint, const float4& distortion )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	gpuIntrinsicWarp<<<gridDim, blockDim>>>(input, output, width, height,
									focalLength, principalPoint,
									distortion.x, distortion.y, distortion.z, distortion.w);

	return CUDA(hipGetLastError());
}


// cudaWarpIntrinsic
hipError_t cudaWarpIntrinsic( float4* input, float4* output, uint32_t width, uint32_t height,
						 const float2& focalLength, const float2& principalPoint, const float4& distortion )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	gpuIntrinsicWarp<<<gridDim, blockDim>>>(input, output, width, height,
									focalLength, principalPoint,
									distortion.x, distortion.y, distortion.z, distortion.w);

	return CUDA(hipGetLastError());
}






