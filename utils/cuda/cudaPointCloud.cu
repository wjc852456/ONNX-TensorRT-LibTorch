#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaPointCloud.h"
#include "cudaColormap.h"

#include "logging.h"


// float4 to uchar3
inline __host__ __device__ uchar3 make_uchar3( float4 a )
{
    return make_uchar3(fminf(a.x, 255.0f), fminf(a.y, 255.0f), fminf(a.z, 255.0f));
}


// gpuPointCloudExtract
template<bool useRGB>
__global__ void gpuPointCloudExtract( float* depth, float4* rgba, int width, int height, 
							   float2 fx, float2 cx, cudaPointCloud::Vertex* points )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int i = y * width + x;

	if( x >= width || y >= height )
		return;

	// read depth map sample
	const float depth_sample = depth[i];

	// create output point
	cudaPointCloud::Vertex point;

	point.classID = 0;	// not applied yet

	// apply depth calibration
	point.pos = make_float3((float(x) - cx.x) * depth_sample / fx.x,
				    	    (float(y) - cx.y) * depth_sample / fx.y * -1.0f,
				     	depth_sample * -1.0f);

	// read RGB if needed
	if( useRGB )
		point.color = make_uchar3(rgba[i]);
	else
		point.color = make_uchar3(255,255,255);

	// save the point
	points[i] = point;
}


// Extract
bool cudaPointCloud::Extract( float* depth, uint32_t depth_width, uint32_t depth_height,
						float4* rgba, uint32_t color_width, uint32_t color_height )
{
	if( !depth )
	{
		LogError(LOG_CUDA "cudaPointCloud::Extract() -- depth map is NULL\n");
		return false;
	}

	if( depth_width == 0 || depth_height == 0 || color_width == 0 || color_height == 0 )
	{
		LogError(LOG_CUDA "cudaPointCloud::Extract() -- depth width/height parameters are zero\n");
		return false;
	}

	if( rgba != NULL && (color_width == 0 || color_height == 0) )
	{
		LogError(LOG_CUDA "cudaPointCloud::Extract() -- color width/height parameters are zero\n");
		return false;
	}

	// determine if RGB used
	if( rgba != NULL )
		mHasRGB = true;

	// upsample depth if needed
	if( mHasRGB && (depth_width != color_width || depth_height != color_height) )
	{
		if( !allocDepthResize(color_width * color_height * sizeof(float)) )
			return false;

		if( CUDA_FAILED(cudaColormap(depth, depth_width, depth_height,
							    mDepthResize, color_width, color_height,
							    make_float2(0.0f,0.0f), COLORMAP_NONE, FILTER_LINEAR)) ) 
		{
			LogError(LOG_CUDA "cudaPointCloud::Extract() -- failed to resize depth image\n");
			return false; 
		}

		depth        = mDepthResize;
		depth_width  = color_width;
		depth_height = color_height;
	}

	// allocate point cloud memory
	const uint32_t numPoints = depth_width * depth_height;

	if( !Reserve(numPoints) )
		return false;
		
	// default calibration if needed
	if( !mHasCalibration )
	{
		const float f_w = (float)depth_width;
		const float f_h = (float)depth_height;

		mFocalLength = make_float2(f_h, f_h);
		mPrincipalPoint = make_float2(f_w * 0.5f, f_h * 0.5f);
	}

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(depth_width,blockDim.x), iDivUp(depth_height,blockDim.y));

	if( mHasRGB )
		gpuPointCloudExtract<true><<<gridDim, blockDim>>>(depth, rgba, depth_width, depth_height, mFocalLength, mPrincipalPoint, mPointsGPU);
	else
		gpuPointCloudExtract<false><<<gridDim, blockDim>>>(depth, rgba, depth_width, depth_height, mFocalLength, mPrincipalPoint, mPointsGPU);

	// check for launch errors
	if( CUDA_FAILED(hipGetLastError()) )
	{
		LogError(LOG_CUDA "cudaPointCloud::Extract() -- failed to extra point cloud with CUDA\n");
		return false;
	}
	
	mNumPoints = numPoints;
	mHasNewPoints = true;

	return true;
}


// Extract
bool cudaPointCloud::Extract( float* depth, float4* rgba, uint32_t width, uint32_t height )
{
	return Extract(depth, width, height, rgba, width, height);
}







