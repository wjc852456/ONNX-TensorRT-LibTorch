#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaYUV.h"
#include "cudaVector.h"

#define COLOR_COMPONENT_MASK            0x3FF
#define COLOR_COMPONENT_BIT_SIZE        10

#define FIXED_DECIMAL_POINT             24
#define FIXED_POINT_MULTIPLIER          1.0f
#define FIXED_COLOR_COMPONENT_MASK      0xffffffff


//-----------------------------------------------------------------------------------
// YUV to RGB colorspace conversion
//-----------------------------------------------------------------------------------
static inline __device__ float clamp( float x )	{ return fminf(fmaxf(x, 0.0f), 255.0f); }

// YUV2RGB
template<typename T>
static inline __device__ T YUV2RGB(const uint3& yuvi)
{
	const float luma = float(yuvi.x);
	const float u    = float(yuvi.y) - 512.0f;
	const float v    = float(yuvi.z) - 512.0f;
	const float s    = 1.0f / 1024.0f * 255.0f;	// TODO clamp for uchar output?

#if 1
	return make_vec<T>(clamp((luma + 1.402f * v) * s),
				    clamp((luma - 0.344f * u - 0.714f * v) * s),
				    clamp((luma + 1.772f * u) * s), 255);
#else
	return make_vec<T>(clamp((luma + 1.140f * v) * s),
				    clamp((luma - 0.395f * u - 0.581f * v) * s),
				    clamp((luma + 2.032f * u) * s), 255);
#endif
}


//-----------------------------------------------------------------------------------
// NV12 to RGB
//-----------------------------------------------------------------------------------
template<typename T>
__global__ void NV12ToRGB(uint32_t* srcImage, size_t nSourcePitch,
                          T* dstImage,        size_t nDestPitch,
                          uint32_t width,     uint32_t height)
{
	int x, y;
	uint32_t yuv101010Pel[2];
	uint32_t processingPitch = ((width) + 63) & ~63;
	uint8_t *srcImageU8     = (uint8_t *)srcImage;

	processingPitch = nSourcePitch;

	// Pad borders with duplicate pixels, and we multiply by 2 because we process 2 pixels per thread
	x = blockIdx.x * (blockDim.x << 1) + (threadIdx.x << 1);
	y = blockIdx.y *  blockDim.y       +  threadIdx.y;

	if( x >= width )
		return; //x = width - 1;

	if( y >= height )
		return; // y = height - 1;

	// Read 2 Luma components at a time, so we don't waste processing since CbCr are decimated this way.
	// if we move to texture we could read 4 luminance values
	yuv101010Pel[0] = (srcImageU8[y * processingPitch + x    ]) << 2;
	yuv101010Pel[1] = (srcImageU8[y * processingPitch + x + 1]) << 2;

	uint32_t chromaOffset    = processingPitch * height;
	int y_chroma = y >> 1;

	if (y & 1)  // odd scanline ?
	{
		uint32_t chromaCb;
		uint32_t chromaCr;

		chromaCb = srcImageU8[chromaOffset + y_chroma * processingPitch + x    ];
		chromaCr = srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1];

		if (y_chroma < ((height >> 1) - 1)) // interpolate chroma vertically
		{
			chromaCb = (chromaCb + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x    ] + 1) >> 1;
			chromaCr = (chromaCr + srcImageU8[chromaOffset + (y_chroma + 1) * processingPitch + x + 1] + 1) >> 1;
		}

		yuv101010Pel[0] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[0] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

		yuv101010Pel[1] |= (chromaCb << (COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[1] |= (chromaCr << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
	}
	else
	{
		yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[0] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));

		yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x    ] << (COLOR_COMPONENT_BIT_SIZE       + 2));
		yuv101010Pel[1] |= ((uint32_t)srcImageU8[chromaOffset + y_chroma * processingPitch + x + 1] << ((COLOR_COMPONENT_BIT_SIZE << 1) + 2));
	}

	// this steps performs the color conversion
	const uint3 yuvi_0 = make_uint3((yuv101010Pel[0] &   COLOR_COMPONENT_MASK),
	                               ((yuv101010Pel[0] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK),
					               ((yuv101010Pel[0] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK));
  
	const uint3 yuvi_1 = make_uint3((yuv101010Pel[1] &   COLOR_COMPONENT_MASK),
							       ((yuv101010Pel[1] >>  COLOR_COMPONENT_BIT_SIZE)       & COLOR_COMPONENT_MASK),
								   ((yuv101010Pel[1] >> (COLOR_COMPONENT_BIT_SIZE << 1)) & COLOR_COMPONENT_MASK));
								   
	// YUV to RGB transformation conversion
	dstImage[y * width + x]     = YUV2RGB<T>(yuvi_0);
	dstImage[y * width + x + 1] = YUV2RGB<T>(yuvi_1);
}


template<typename T> 
static hipError_t launchNV12ToRGB( void* srcDev, T* dstDev, size_t width, size_t height )
{
	if( !srcDev || !dstDev )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0 )
		return hipErrorInvalidValue;

	const size_t srcPitch = width * sizeof(uint8_t);
	const size_t dstPitch = width * sizeof(T);
	
	const dim3 blockDim(32,8,1);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height, blockDim.y), 1);

	NV12ToRGB<T><<<gridDim, blockDim>>>( (uint32_t*)srcDev, srcPitch, dstDev, dstPitch, width, height );
	
	return CUDA(hipGetLastError());
}

// cudaNV12ToRGB (uchar3)
hipError_t cudaNV12ToRGB( void* srcDev, uchar3* destDev, size_t width, size_t height )
{
	return launchNV12ToRGB<uchar3>(srcDev, destDev, width, height);
}

// cudaNV12ToRGB (float3)
hipError_t cudaNV12ToRGB( void* srcDev, float3* destDev, size_t width, size_t height )
{
	return launchNV12ToRGB<float3>(srcDev, destDev, width, height);
}

// cudaNV12ToRGBA (uchar4)
hipError_t cudaNV12ToRGBA( void* srcDev, uchar4* destDev, size_t width, size_t height )
{
	return launchNV12ToRGB<uchar4>(srcDev, destDev, width, height);
}

// cudaNV12ToRGBA (float4)
hipError_t cudaNV12ToRGBA( void* srcDev, float4* destDev, size_t width, size_t height )
{
	return launchNV12ToRGB<float4>(srcDev, destDev, width, height);
}


#if 0
// cudaNV12SetupColorspace
hipError_t cudaNV12SetupColorspace( float hue )
{
	const float hueSin = sin(hue);
	const float hueCos = cos(hue);

	float hueCSC[9];

	const bool itu601 = false;

	if( itu601 /*CSC == ITU601*/)
	{
		//CCIR 601
		hueCSC[0] = 1.1644f;
		hueCSC[1] = hueSin * 1.5960f;
		hueCSC[2] = hueCos * 1.5960f;
		hueCSC[3] = 1.1644f;
		hueCSC[4] = (hueCos * -0.3918f) - (hueSin * 0.8130f);
		hueCSC[5] = (hueSin *  0.3918f) - (hueCos * 0.8130f);
		hueCSC[6] = 1.1644f;
		hueCSC[7] = hueCos *  2.0172f;
		hueCSC[8] = hueSin * -2.0172f;
	}
	else /*if(CSC == ITU709)*/
	{
		//CCIR 709
		hueCSC[0] = 1.0f;
		hueCSC[1] = hueSin * 1.57480f;
		hueCSC[2] = hueCos * 1.57480f;
		hueCSC[3] = 1.0;
		hueCSC[4] = (hueCos * -0.18732f) - (hueSin * 0.46812f);
		hueCSC[5] = (hueSin *  0.18732f) - (hueCos * 0.46812f);
		hueCSC[6] = 1.0f;
		hueCSC[7] = hueCos *  1.85560f;
		hueCSC[8] = hueSin * -1.85560f;
	}


	if( CUDA_FAILED(hipMemcpyToSymbol(HIP_SYMBOL(constHueColorSpaceMat), hueCSC, sizeof(float) * 9)) )
		return hipErrorInvalidSymbol;

	uint32_t cudaAlpha = ((uint32_t)0xff<< 24);

	if( CUDA_FAILED(hipMemcpyToSymbol(HIP_SYMBOL(constAlpha), &cudaAlpha, sizeof(uint32_t))) )
		return hipErrorInvalidSymbol;

	nv12ColorspaceSetup = true;
	return hipSuccess;
}
#endif

