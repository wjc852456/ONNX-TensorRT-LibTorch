#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaWarp.h"


// cudaFisheye
template<typename T>
__global__ void cudaFisheye( T* input, T* output, int width, int height, float focus)
{
	const int2 uv_out = make_int2(blockDim.x * blockIdx.x + threadIdx.x,
				               blockDim.y * blockIdx.y + threadIdx.y);
						   
	if( uv_out.x >= width || uv_out.y >= height )
		return;
	
	const float fWidth  = width;
	const float fHeight = height;
	
	// convert to cartesian coordinates
	const float cx = ((uv_out.x / fWidth) - 0.5f)  * 2.0f;	
	const float cy = (0.5f - (uv_out.y / fHeight)) * 2.0f;

	const float theta = atan2f(cy, cx);
	const float r     = atanf(sqrtf(cx*cx+cy*cy) * focus);
	
	const float tx = r * __cosf(theta);
	const float ty = r * __sinf(theta);
	
	// convert back out of cartesian coordinates
	float u = (tx * 0.5f + 0.5f) * fWidth;
	float v = (0.5f - (ty * 0.5f)) * fHeight;

	if( u < 0.0f ) u = 0.0f;
	if( v < 0.0f ) v = 0.0f;

	if( u > fWidth  - 1.0f ) u = fWidth - 1.0f;
	if( v > fHeight - 1.0f ) v = fHeight - 1.0f;
	
	output[uv_out.y * width + uv_out.x] = input[(int)v * width + (int)u];
} 


// cudaWarpFisheye
hipError_t cudaWarpFisheye( uchar4* input, uchar4* output, uint32_t width, uint32_t height, float focus )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	cudaFisheye<<<gridDim, blockDim>>>(input, output, width, height, focus);

	return CUDA(hipGetLastError());
}


// cudaWarpFisheye
hipError_t cudaWarpFisheye( float4* input, float4* output, uint32_t width, uint32_t height, float focus )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( width == 0 || height == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	cudaFisheye<<<gridDim, blockDim>>>(input, output, width, height, focus);

	return CUDA(hipGetLastError());
}


