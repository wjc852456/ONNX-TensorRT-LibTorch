#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2019, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaCrop.h"



// gpuCrop
template<typename T>
__global__ void gpuCrop( T* input, T* output, int offsetX, int offsetY, 
					int inWidth, int outWidth, int outHeight )
{
	const int out_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int out_y = blockIdx.y * blockDim.y + threadIdx.y;

	if( out_x >= outWidth || out_y >= outHeight )
		return;

	const int in_x = out_x + offsetX;
	const int in_y = out_y + offsetY;

	output[out_y * outWidth + out_x] = input[in_y * inWidth + in_x];
}


// launchCrop
template<typename T>
static hipError_t launchCrop( T* input, T* output, const int4& roi, size_t inputWidth, size_t inputHeight )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || inputHeight == 0 )
		return hipErrorInvalidValue;

	// get the ROI/output dimensions
	const int outputWidth = roi.z - roi.x;
	const int outputHeight = roi.w - roi.y;

	// validate the requested ROI
	if( outputWidth <= 0 || outputHeight <= 0 )
		return hipErrorInvalidValue;

	if( outputWidth > inputWidth || outputHeight > inputHeight )
		return hipErrorInvalidValue;

	if( roi.x < 0 || roi.y < 0 || roi.z < 0 || roi.w < 0 )
		return hipErrorInvalidValue;

	if( roi.z >= inputWidth || roi.w >= inputHeight )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuCrop<T><<<gridDim, blockDim>>>(input, output, roi.x, roi.y, inputWidth, outputWidth, outputHeight);

	return CUDA(hipGetLastError());
}

// cudaCrop (uint8 grayscale)
hipError_t cudaCrop( uint8_t* input, uint8_t* output, const int4& roi, size_t inputWidth, size_t inputHeight )
{
	return launchCrop<uint8_t>(input, output, roi, inputWidth, inputHeight);
}

// cudaCrop (float grayscale)
hipError_t cudaCrop( float* input, float* output, const int4& roi, size_t inputWidth, size_t inputHeight )
{
	return launchCrop<float>(input, output, roi, inputWidth, inputHeight);
}

// cudaCrop (uchar3)
hipError_t cudaCrop( uchar3* input, uchar3* output, const int4& roi, size_t inputWidth, size_t inputHeight )
{
	return launchCrop<uchar3>(input, output, roi, inputWidth, inputHeight);
}

// cudaCrop (uchar4)
hipError_t cudaCrop( uchar4* input, uchar4* output, const int4& roi, size_t inputWidth, size_t inputHeight )
{
	return launchCrop<uchar4>(input, output, roi, inputWidth, inputHeight);
}

// cudaCrop (float3)
hipError_t cudaCrop( float3* input, float3* output, const int4& roi, size_t inputWidth, size_t inputHeight )
{
	return launchCrop<float3>(input, output, roi, inputWidth, inputHeight);
}

// cudaCrop (float4)
hipError_t cudaCrop( float4* input, float4* output, const int4& roi, size_t inputWidth, size_t inputHeight )
{
	return launchCrop<float4>(input, output, roi, inputWidth, inputHeight);
}

//-----------------------------------------------------------------------------------
hipError_t cudaCrop( void* input, void* output, const int4& roi, size_t inputWidth, size_t inputHeight, imageFormat format )
{
	if( format == IMAGE_RGB8 || format == IMAGE_BGR8 )
		return cudaCrop((uchar3*)input, (uchar3*)output, roi, inputWidth, inputHeight);
	else if( format == IMAGE_RGBA8 || format == IMAGE_BGRA8 )
		return cudaCrop((uchar4*)input, (uchar4*)output, roi, inputWidth, inputHeight);
	else if( format == IMAGE_RGB32F || format == IMAGE_BGR32F )
		return cudaCrop((float3*)input, (float3*)output, roi, inputWidth, inputHeight);
	else if( format == IMAGE_RGBA32F || format == IMAGE_BGRA32F )
		return cudaCrop((float4*)input, (float4*)output, roi, inputWidth, inputHeight);
	else if( format == IMAGE_GRAY8 )
		return cudaCrop((uint8_t*)input, (uint8_t*)output, roi, inputWidth, inputHeight);
	else if( format == IMAGE_GRAY32F )
		return cudaCrop((float*)input, (float*)output, roi, inputWidth, inputHeight);

	LogError(LOG_CUDA "cudaCrop() -- invalid image format '%s'\n", imageFormatToStr(format));
	LogError(LOG_CUDA "              supported formats are:\n");
	LogError(LOG_CUDA "                  * gray8\n");
	LogError(LOG_CUDA "                  * gray32f\n");
	LogError(LOG_CUDA "                  * rgb8, bgr8\n");
	LogError(LOG_CUDA "                  * rgba8, bgra8\n");
	LogError(LOG_CUDA "                  * rgb32f, bgr32f\n");
	LogError(LOG_CUDA "                  * rgba32f, bgra32f\n");

	return hipErrorInvalidValue;
}




