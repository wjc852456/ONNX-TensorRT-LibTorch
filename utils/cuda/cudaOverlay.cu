#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaOverlay.h"


// cudaOverlay
template<typename T>
__global__ void gpuOverlay( T* input, int inputWidth, int inputHeight, T* output, int outputWidth, int outputHeight, int x0, int y0 ) 
{
	const int input_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int input_y = blockIdx.y * blockDim.y + threadIdx.y;

	const int x = input_x + x0;
	const int y = input_y + y0;
	
	if( input_x >= inputWidth || input_y >= inputHeight || x >= outputWidth || y >= outputHeight )
		return;

	output[y * outputWidth + x] = input[input_y * inputWidth + input_x];
}

hipError_t cudaOverlay( void* input, size_t inputWidth, size_t inputHeight,
						 void* output, size_t outputWidth, size_t outputHeight,
						 imageFormat format, int x, int y )
{
	if( !input || !output || inputWidth == 0 || inputHeight == 0 || outputWidth == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;
	
	if( x < 0 || y < 0 || x >= outputWidth || y >= outputHeight )
		return hipErrorInvalidValue;
	
	if( !imageFormatIsRGB(format) && !imageFormatIsBGR(format) && !imageFormatIsGray(format) )
		return hipErrorInvalidValue;
	
	int overlayWidth = inputWidth;
	int overlayHeight = inputHeight;

	if( x + overlayWidth >= outputWidth )
		overlayWidth = outputWidth - x;

	if( y + overlayHeight >= outputHeight )
		overlayHeight = outputHeight - y;
	
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(overlayWidth,blockDim.x), iDivUp(overlayHeight,blockDim.y));

	#define launch_overlay(type)	\
		gpuOverlay<type><<<gridDim, blockDim>>>((type*)input, inputWidth, inputHeight, (type*)output, outputWidth, outputHeight, x, y)
	
	if( format == IMAGE_RGB8 || format == IMAGE_BGR8 )
		launch_overlay(uchar3);
	else if( format == IMAGE_RGBA8 || format == IMAGE_BGRA8 )
		launch_overlay(uchar4);
	else if( format == IMAGE_RGB32F || format == IMAGE_BGR32F )
		launch_overlay(float3);
	else if( format == IMAGE_RGBA32F || format == IMAGE_BGRA32F )
		launch_overlay(float4);
	else if( format == IMAGE_GRAY8 )
		launch_overlay(uint8_t);
	else if( format == IMAGE_GRAY32F )
		launch_overlay(float);
	
	return hipGetLastError();
}	
							 
							 
//----------------------------------------------------------------------------						 
template<typename T>
__global__ void gpuRectFill( T* input, T* output, int width, int height,
					    float4* rects, int numRects, float4 color ) 
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= width || y >= height )
		return;

	T px = input[ y * width + x ];

	const float fx = x;
	const float fy = y;
	
	const float alpha = color.w / 255.0f;
	const float ialph = 1.0f - alpha;
	
	for( int nr=0; nr < numRects; nr++ )
	{
		const float4 r = rects[nr];
	
		if( fy >= r.y && fy <= r.w && fx >= r.x && fx <= r.z )
		{
			px.x = alpha * color.x + ialph * px.x;
			px.y = alpha * color.y + ialph * px.y;
			px.z = alpha * color.z + ialph * px.z;
		}
	}
	
	output[y * width + x] = px;	 
}

template<typename T>
__global__ void gpuRectFillBox( T* input, T* output, int imgWidth, int imgHeight, int x0, int y0, int boxWidth, int boxHeight, const float4 color ) 
{
	const int box_x = blockIdx.x * blockDim.x + threadIdx.x;
	const int box_y = blockIdx.y * blockDim.y + threadIdx.y;

	if( box_x >= boxWidth || box_y >= boxHeight )
		return;

	const int x = box_x + x0;
	const int y = box_y + y0;

	if( x >= imgWidth || y >= imgHeight )
		return;

	T px = input[ y * imgWidth + x ];

	const float alpha = color.w / 255.0f;
	const float ialph = 1.0f - alpha;

	px.x = alpha * color.x + ialph * px.x;
	px.y = alpha * color.y + ialph * px.y;
	px.z = alpha * color.z + ialph * px.z;
	
	output[y * imgWidth + x] = px;
}

template<typename T>
hipError_t launchRectFill( T* input, T* output, size_t width, size_t height, float4* rects, int numRects, const float4& color )
{
	if( !input || !output || width == 0 || height == 0 || !rects || numRects == 0 )
		return hipErrorInvalidValue;

	// if input and output are the same image, then we can use the faster method
	// which draws 1 box per kernel, but doesn't copy pixels that aren't inside boxes
	if( input == output )
	{
		for( int n=0; n < numRects; n++ )
		{
			const int boxWidth = (int)(rects[n].z - rects[n].x);
			const int boxHeight = (int)(rects[n].w - rects[n].y);

			// launch kernel
			const dim3 blockDim(8, 8);
			const dim3 gridDim(iDivUp(boxWidth,blockDim.x), iDivUp(boxHeight,blockDim.y));

			gpuRectFillBox<T><<<gridDim, blockDim>>>(input, output, width, height, (int)rects[n].x, (int)rects[n].y, boxWidth, boxHeight, color); 
		}
	}
	else
	{
		// launch kernel
		const dim3 blockDim(8, 8);
		const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

		gpuRectFill<T><<<gridDim, blockDim>>>(input, output, width, height, rects, numRects, color);
	}

	return hipGetLastError();
}

// cudaRectFill
hipError_t cudaRectFill( void* input, void* output, size_t width, size_t height, imageFormat format, float4* rects, int numRects, const float4& color )
{
	if( !input || !output || width == 0 || height == 0 || !rects || numRects == 0 )
		return hipErrorInvalidValue;

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(width,blockDim.x), iDivUp(height,blockDim.y));

	if( format == IMAGE_RGB8 )
		return launchRectFill<uchar3>((uchar3*)input, (uchar3*)output, width, height, rects, numRects, color); 
	else if( format == IMAGE_RGBA8 )
		return launchRectFill<uchar4>((uchar4*)input, (uchar4*)output, width, height, rects, numRects, color); 
	else if( format == IMAGE_RGB32F )
		return launchRectFill<float3>((float3*)input, (float3*)output, width, height, rects, numRects, color); 
	else if( format == IMAGE_RGBA32F )
		return launchRectFill<float4>((float4*)input, (float4*)output, width, height, rects, numRects, color); 
	else
		return hipErrorInvalidValue;
}
