#include "hip/hip_runtime.h"
/*
 * Copyright (c) 2017, NVIDIA CORPORATION. All rights reserved.
 *
 * Permission is hereby granted, free of charge, to any person obtaining a
 * copy of this software and associated documentation files (the "Software"),
 * to deal in the Software without restriction, including without limitation
 * the rights to use, copy, modify, merge, publish, distribute, sublicense,
 * and/or sell copies of the Software, and to permit persons to whom the
 * Software is furnished to do so, subject to the following conditions:
 *
 * The above copyright notice and this permission notice shall be included in
 * all copies or substantial portions of the Software.
 *
 * THE SOFTWARE IS PROVIDED "AS IS", WITHOUT WARRANTY OF ANY KIND, EXPRESS OR
 * IMPLIED, INCLUDING BUT NOT LIMITED TO THE WARRANTIES OF MERCHANTABILITY,
 * FITNESS FOR A PARTICULAR PURPOSE AND NONINFRINGEMENT.  IN NO EVENT SHALL
 * THE AUTHORS OR COPYRIGHT HOLDERS BE LIABLE FOR ANY CLAIM, DAMAGES OR OTHER
 * LIABILITY, WHETHER IN AN ACTION OF CONTRACT, TORT OR OTHERWISE, ARISING
 * FROM, OUT OF OR IN CONNECTION WITH THE SOFTWARE OR THE USE OR OTHER
 * DEALINGS IN THE SOFTWARE.
 */

#include "cudaYUV.h"
#include "imageFormat.h"


//-----------------------------------------------------------------------------------
// YUV to RGB colorspace conversion
//-----------------------------------------------------------------------------------
static inline __device__ float clamp( float x )	
{ 
	return fminf(fmaxf(x, 0.0f), 255.0f); 
}

static inline __device__ float3 YUV2RGB(float Y, float U, float V)
{
	U -= 128.0f;
	V -= 128.0f;

#if 1
	return make_float3(clamp(Y + 1.4065f * V),
    				    clamp(Y - 0.3455f * U - 0.7169f * V),
				    clamp(Y + 1.7790f * U));
#else
	return make_float3(clamp(Y + 1.402f * V),
    				    clamp(Y - 0.344f * U - 0.714f * V),
				    clamp(Y + 1.772f * U));
#endif
}

//-----------------------------------------------------------------------------------
// YUYV/UYVY are macropixel formats, and two RGB pixels are output at once.
// Define vectors with 6 and 8 elements so they can be written at one time.
// These are similar to those from cudaVector.h, except for 6/8 elements.
//-----------------------------------------------------------------------------------
struct /*__align__(6)*/ uchar6
{
   uint8_t x0, y0, z0, x1, y1, z1;
};

struct __align__(8) uchar8
{
   uint8_t x0, y0, z0, w0, x1, y1, z1, w1;
};

struct /*__align__(24)*/ float6
{
   float x0, y0, z0, x1, y1, z1;
};

struct __align__(32) float8
{
   float x0, y0, z0, w0, x1, y1, z1, w1;
};

template<class T> struct vecTypeInfo;

template<> struct vecTypeInfo<uchar6> { typedef uint8_t Base; };
template<> struct vecTypeInfo<uchar8> { typedef uint8_t Base; };

template<> struct vecTypeInfo<float6> { typedef float Base; };
template<> struct vecTypeInfo<float8> { typedef float Base; };

template<typename T> struct vec_assert_false : std::false_type { };

#define BaseType typename vecTypeInfo<T>::Base

template<typename T> inline __host__ __device__ T make_vec(BaseType x0, BaseType y0, BaseType z0, BaseType w0, BaseType x1, BaseType y1, BaseType z1, BaseType w1) { static_assert(vec_assert_false<T>::value, "invalid vector type - supported types are uchar6, uchar8, float6, float8");  }

template<> inline __host__ __device__ uchar6 make_vec( uint8_t x0, uint8_t y0, uint8_t z0, uint8_t w0, uint8_t x1, uint8_t y1, uint8_t z1, uint8_t w1 )	{ return {x0, y0, z0, x1, y1, z1}; }
template<> inline __host__ __device__ uchar8 make_vec( uint8_t x0, uint8_t y0, uint8_t z0, uint8_t w0, uint8_t x1, uint8_t y1, uint8_t z1, uint8_t w1 )	{ return {x0, y0, z0, w1, x1, y1, z1, w1}; }

template<> inline __host__ __device__ float6 make_vec( float x0, float y0, float z0, float w0, float x1, float y1, float z1, float w1 )				{ return {x0, y0, z0, x1, y1, z1}; }
template<> inline __host__ __device__ float8 make_vec( float x0, float y0, float z0, float w0, float x1, float y1, float z1, float w1 )				{ return {x0, y0, z0, w1, x1, y1, z1, w1}; }


//-----------------------------------------------------------------------------------
// YUYV/UYVY to RGBA
//-----------------------------------------------------------------------------------
template <typename T, imageFormat format>
__global__ void YUYVToRGBA( uchar4* src, T* dst, int halfWidth, int height )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;

	if( x >= halfWidth || y >= height )
		return;

	const uchar4 macroPx = src[y * halfWidth + x];

	// Y0 is the brightness of pixel 0, Y1 the brightness of pixel 1.
	// U and V is the color of both pixels.
	float y0, y1, u, v;

	if( format == IMAGE_YUYV )
	{
		// YUYV [ Y0 | U0 | Y1 | V0 ]
		y0 = macroPx.x;
		y1 = macroPx.z;
		u  = macroPx.y;
		v  = macroPx.w;
	}
	else if( format == IMAGE_YVYU )
	{
		// YVYU [ Y0 | V0 | Y1 | U0 ]
		y0 = macroPx.x;
		y1 = macroPx.z;
		u  = macroPx.w;
		v  = macroPx.y;
	}
	else // if( format == IMAGE_UYVY )
	{
		// UYVY [ U0 | Y0 | V0 | Y1 ]
		y0 = macroPx.y;
		y1 = macroPx.w;
		u  = macroPx.x;
		v  = macroPx.z;
	}

	// this function outputs two pixels from one YUYV macropixel
	const float3 px0 = YUV2RGB(y0, u, v);
	const float3 px1 = YUV2RGB(y1, u, v);

	dst[y * halfWidth + x] = make_vec<T>(px0.x, px0.y, px0.z, 255,
								  px1.x, px1.y, px1.z, 255);
} 

template<typename T, imageFormat format>
static hipError_t launchYUYVToRGB( void* input, T* output, size_t width, size_t height)
{
	if( !input || !output || !width || !height )
		return hipErrorInvalidValue;

	const int  halfWidth = width / 2;	// two pixels are output at once
	const dim3 blockDim(8,8);
	const dim3 gridDim(iDivUp(halfWidth, blockDim.x), iDivUp(height, blockDim.y));

	YUYVToRGBA<T, format><<<gridDim, blockDim>>>((uchar4*)input, output, halfWidth, height);

	return CUDA(hipGetLastError());
}


// cudaYUYVToRGB (uchar3)
hipError_t cudaYUYVToRGB( void* input, uchar3* output, size_t width, size_t height )
{
	return launchYUYVToRGB<uchar6, IMAGE_YUYV>(input, (uchar6*)output, width, height);
}

// cudaYUYVToRGB (float3)
hipError_t cudaYUYVToRGB( void* input, float3* output, size_t width, size_t height )
{
	return launchYUYVToRGB<float6, IMAGE_YUYV>(input, (float6*)output, width, height);
}

// cudaYUYVToRGBA (uchar4)
hipError_t cudaYUYVToRGBA( void* input, uchar4* output, size_t width, size_t height )
{
	return launchYUYVToRGB<uchar8, IMAGE_YUYV>(input, (uchar8*)output, width, height);
}

// cudaYUYVToRGBA (float4)
hipError_t cudaYUYVToRGBA( void* input, float4* output, size_t width, size_t height )
{
	return launchYUYVToRGB<float8, IMAGE_YUYV>(input, (float8*)output, width, height);
}

//-----------------------------------------------------------------------------------

// cudaUYVYToRGB (uchar3)
hipError_t cudaUYVYToRGB( void* input, uchar3* output, size_t width, size_t height )
{
	return launchYUYVToRGB<uchar6, IMAGE_UYVY>(input, (uchar6*)output, width, height);
}

// cudaUYVYToRGB (float3)
hipError_t cudaUYVYToRGB( void* input, float3* output, size_t width, size_t height )
{
	return launchYUYVToRGB<float6, IMAGE_UYVY>(input, (float6*)output, width, height);
}

// cudaUYVYToRGBA (uchar4)
hipError_t cudaUYVYToRGBA( void* input, uchar4* output, size_t width, size_t height )
{
	return launchYUYVToRGB<uchar8, IMAGE_UYVY>(input, (uchar8*)output, width, height);
}

// cudaUYVYToRGBA (float4)
hipError_t cudaUYVYToRGBA( void* input, float4* output, size_t width, size_t height )
{
	return launchYUYVToRGB<float8, IMAGE_UYVY>(input, (float8*)output, width, height);
}

//-----------------------------------------------------------------------------------

// cudaYVYUToRGB (uchar3)
hipError_t cudaYVYUToRGB( void* input, uchar3* output, size_t width, size_t height )
{
	return launchYUYVToRGB<uchar6, IMAGE_YVYU>(input, (uchar6*)output, width, height);
}

// cudaYUYVToRGB (float3)
hipError_t cudaYVYUToRGB( void* input, float3* output, size_t width, size_t height )
{
	return launchYUYVToRGB<float6, IMAGE_YVYU>(input, (float6*)output, width, height);
}

// cudaYUYVToRGBA (uchar4)
hipError_t cudaYVYUToRGBA( void* input, uchar4* output, size_t width, size_t height )
{
	return launchYUYVToRGB<uchar8, IMAGE_YVYU>(input, (uchar8*)output, width, height);
}

// cudaYUYVToRGBA (float4)
hipError_t cudaYVYUToRGBA( void* input, float4* output, size_t width, size_t height )
{
	return launchYUYVToRGB<float8, IMAGE_YVYU>(input, (float8*)output, width, height);
}

